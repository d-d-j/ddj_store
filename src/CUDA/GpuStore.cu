#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../Store/storeElement.h"
#include "../Store/infoElement.h"
#include "../Store/LoggerHelper.h"

#define DEBUG 1
#define MB_SIZE 1048576
#define CHECK_CUDA_ERR(sth) { gpuAssert((sth), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

extern "C"
{
	int gpuGetCudaDevicesCount()
	{
		int count = 0;
		CHECK_CUDA_ERR( hipGetDeviceCount(&count) );
		return count;
	}

	void gpuGetMemoryUsage(size_t* freeMemory, size_t* totalMemory)
	{
		CHECK_CUDA_ERR( hipMemGetInfo(freeMemory, totalMemory) );

		#if DEBUG
			fprintf(stdout,
					"Free memory: %.2f MB Total memory: %.2f MB File[%s] Line[%d]\n",
					(float)*freeMemory/MB_SIZE,
					(float)*totalMemory/MB_SIZE,
					__FILE__,
					__LINE__);
		#endif
	}

	int gpuAllocateMainArray(size_t size, void** array)
	{
		size_t freeMemory, totalMemory;

		#if DEBUG
		{
			fprintf(stdout,
					"Attempt to allocate %.2f MB memory on GPU File[%s] Line[%d]\n",
					(float)size/MB_SIZE,
					__FILE__,
					__LINE__);
			gpuGetMemoryUsage(&freeMemory, &totalMemory);
		}
		#endif

		hipError_t result = hipSuccess;

		if(totalMemory <= size)
		{
			result = hipErrorOutOfMemory;
			fprintf(stderr, "Size of memory to allocate is bigger than total gpu memory\n");
			return result;
		}

		result = hipMalloc((void**)array, size);

		#if DEBUG
		{
			if(result == hipSuccess)
				fprintf(stdout, "Main gpu array allocated successfully\n");
			else
				fprintf(stderr, "Main gpu array allocation failed! - %s\n", hipGetErrorString(result));
			gpuGetMemoryUsage(&freeMemory, &totalMemory);
		}
		#endif

		return result;
	}

	void gpuFreeMemory(void* devPtr)
	{
		size_t freeMemory, totalMemory;
		#if DEBUG
			fprintf(stdout, "Releasing gpu pointer\n");
		#endif
		CHECK_CUDA_ERR( hipFree(devPtr) );
		gpuGetMemoryUsage(&freeMemory, &totalMemory);
	}

}	/* extern "C" */
