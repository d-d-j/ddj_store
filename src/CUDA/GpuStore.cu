#include <hip/hip_runtime.h>

extern "C"
{
	int gpuGetCudaDevicesCount()
	{
		int count = 0;
		hipGetDeviceCount(&count);
		return count;
	}

	void gpuGetMemoryUsage(size_t* freeMemory, size_t* totalMemory)
	{
		hipMemGetInfo(freeMemory, totalMemory);
	}

	int gpuAllocateMainArray(size_t size, void** array)
	{
		size_t freeMemory, totalMemory;

		gpuGetMemoryUsage(&freeMemory, &totalMemory);

		hipError_t result = hipSuccess;

		if(totalMemory <= size)
		{
			result = hipErrorOutOfMemory;
			return result;
		}

		result = hipMalloc((void**)array, size);

		gpuGetMemoryUsage(&freeMemory, &totalMemory);

		return result;
	}

	void gpuFreeMemory(void* devPtr)
	{
		size_t freeMemory, totalMemory;
		hipFree(devPtr);
		gpuGetMemoryUsage(&freeMemory, &totalMemory);
	}

}	/* extern "C" */
