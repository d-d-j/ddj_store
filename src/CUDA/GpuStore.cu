#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../Store/storeSettings.h"

extern "C"
{
	int gpuGetCudaDevicesCount()
	{
		int count = 0;
		hipGetDeviceCount(&count);
		return count;
	}

	void gpuGetMemoryUsage(size_t* freeMemory, size_t* totalMemory)
	{
		hipMemGetInfo(freeMemory, totalMemory);
	}

	int gpuAllocateMainArray(size_t size, void** array)
	{
		size_t freeMemory, totalMemory;

		gpuGetMemoryUsage(&freeMemory, &totalMemory);

		hipError_t result = hipSuccess;

		if(totalMemory <= size)
		{
			result = hipErrorOutOfMemory;
			return result;
		}

		result = hipMalloc((void**)array, size);

		gpuGetMemoryUsage(&freeMemory, &totalMemory);

		return result;
	}

	void gpuFreeMemory(void* devPtr)
	{
		size_t freeMemory, totalMemory;
		hipFree(devPtr);
		gpuGetMemoryUsage(&freeMemory, &totalMemory);
	}

	int gpuGetCudaDevicesCountAndPrint()
	{
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;
		int driverVersion = 0, runtimeVersion = 0;

		printf("[File:%s][Line:%d] ==> CUDA : Found %d CUDA devices: \n\n", __FILE__, __LINE__, nDevices);
		for (int i = 0; i < nDevices; i++)
		{
			hipSetDevice(i);
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);
			hipGetDeviceProperties(&prop, i);

			// Log device query
			printf("	Device Number: %d\n", i);
			printf("	Device name: %s\n", prop.name);
	        printf("	CUDA Capability Major/Minor version number:    %d.%d\n", prop.major, prop.minor);
	        printf("	CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
			printf("	Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
			printf("	Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
			printf("	Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		}
		return nDevices;
	}

	bool gpuCheckCudaDevice(int n)
	{
		int driverVersion = 0, runtimeVersion = 0;
		hipDeviceProp_t prop;
		hipSetDevice(n);
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		hipGetDeviceProperties(&prop, n);

		if(prop.major < MIN_CUDA_MAJOR_VER) return false;
		if(prop.minor < MIN_CUDA_MINOR_VER) return false;

		return true;
	}

}	/* extern "C" */
