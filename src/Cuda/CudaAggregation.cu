#include "hip/hip_runtime.h"
#include "CudaQuery.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>

// HOW TO PRINT STH TO CONSOLE IN KERNEL
/*
// System includes
#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
                                  blockIdx.y*gridDim.x+blockIdx.x,\
                                  threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
                                  __VA_ARGS__)
// CUPRINTF("\tIdx: %d, tag: %d, metric: %d, val: %f, Value is:%d\n", idx, tag, elements[idx].metric, elements[idx].value, 1);
*/

// MIN AND MAX

struct min_gpu_elem
{
    __host__ __device__
        storeElement operator()(const storeElement &lhs, const storeElement &rhs) const
    {
    	return lhs.value < rhs.value ? lhs : rhs;
    }
};

struct max_gpu_elem
{
    __host__ __device__
        storeElement operator()(const storeElement &lhs, const storeElement &rhs) const
    {
    	return lhs.value < rhs.value ? rhs : lhs;
    }
};

size_t gpu_max(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	storeElement init;
	hipMemcpy(&init, elements, storeElemSize, hipMemcpyDeviceToHost);

	storeElement* max =
			new storeElement(thrust::reduce(elem_ptr, elem_ptr+elemCount, init, max_gpu_elem()));
	(*result) = max;

	return storeElemSize;
}

size_t gpu_min(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);
	storeElement init;
	hipMemcpy(&init, elements, storeElemSize, hipMemcpyDeviceToHost);

	storeElement* min =
			new storeElement(thrust::reduce(elem_ptr, elem_ptr+elemCount, init, min_gpu_elem()));
	(*result) = min;

	return storeElemSize;
}

// SUM AND AVERAGE

template <typename T>
struct sum_unary_op
{
	__host__ __device__
	float operator()(const T& x) const
	{
		return x.value;
	}
};

size_t gpu_sum(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	float init = 0.0f;
	sum_unary_op<storeElement> unary_op;

	results::sumResult* sum =
			new results::sumResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, thrust::plus<float>()));
	(*result) = sum;

	return sizeof(results::sumResult);
}

size_t gpu_average(ddj::store::storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	float init = 0.0f;
	sum_unary_op<storeElement> unary_op;

	results::averageResult* average =
			new results::averageResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, thrust::plus<float>()), elemCount);
	(*result) = average;

	return sizeof(results::averageResult);
}

// STD DEVIATION AND VARIANCE

template <typename T>
struct variance_unary_op
{
	__host__ __device__
	results::varianceResult operator()(const T& x) const
	{
		results::varianceResult result;
		result.count = 1;
		result.mean = x.value;
		result.M2 = 0;
		return result;
	}
};

struct variance_binary_op
    : public thrust::binary_function<const results::varianceResult&,
                                     const results::varianceResult&,
                                     results::varianceResult >
{
    __host__ __device__
    results::varianceResult operator()(const results::varianceResult& x, const results::varianceResult& y) const
    {
    	results::varianceResult result;

    	float count = x.count + y.count;
    	float delta = y.mean - x.mean;
    	float delta2 = delta * delta;
        result.count = count;
        result.mean = x.mean + delta * y.count / count;
        result.M2 = x.M2 + y.M2;
        result.M2 += delta2 * x.count * y.count / count;

        return result;
    }
};

size_t gpu_stdDeviation(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	variance_unary_op<storeElement> unary_op;
	variance_binary_op binary_op;
	results::varianceResult init;

	results::varianceResult* variance =
			new results::varianceResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, binary_op));
	(*result) = variance;

	return sizeof(results::varianceResult);
}
