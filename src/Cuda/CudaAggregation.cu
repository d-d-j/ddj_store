#include "hip/hip_runtime.h"
#include "CudaQuery.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

struct sum_gpu_elem
{
    __host__ __device__
        gpuElem operator()(const gpuElem &lhs, const gpuElem &rhs) const
    {
    	gpuElem result;
    	result.value = lhs.value+rhs.value;
    	return result;
    }
};

struct min_gpu_elem
{
    __host__ __device__
        gpuElem operator()(const gpuElem &lhs, const gpuElem &rhs) const
    {
    	return lhs.value < rhs.value ? lhs : rhs;
    }
};

struct max_gpu_elem
{
    __host__ __device__
        gpuElem operator()(const gpuElem &lhs, const gpuElem &rhs) const
    {
    	return lhs.value < rhs.value ? rhs : lhs;
    }
};

size_t gpu_add_values(ddj::store::storeElement* elements, size_t dataSize, ddj::store::storeElement** result)
{
	size_t storeElemSize = sizeof(ddj::store::storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<gpuElem> elem_ptr((gpuElem*)elements);
	gpuElem init;
	init.value = 0;
	gpuElem sum = thrust::reduce(elem_ptr, elem_ptr+elemCount, init, sum_gpu_elem());
	hipMalloc(result, storeElemSize);
	hipMemcpy(*result, &sum, storeElemSize, hipMemcpyDeviceToDevice);

	return storeElemSize;
}

size_t gpu_max_from_values(ddj::store::storeElement* elements, size_t dataSize, ddj::store::storeElement** result)
{
	size_t storeElemSize = sizeof(ddj::store::storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<gpuElem> elem_ptr((gpuElem*)elements);
	gpuElem init = elements[0];
	gpuElem max = thrust::reduce(elem_ptr, elem_ptr+elemCount, init, max_gpu_elem());
	hipMalloc(result, storeElemSize);
	hipMemcpy(*result, &max, storeElemSize, hipMemcpyDeviceToDevice);

	return storeElemSize;
}

size_t gpu_min_from_values(ddj::store::storeElement* elements, size_t dataSize, ddj::store::storeElement** result)
{
	size_t storeElemSize = sizeof(ddj::store::storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<gpuElem> elem_ptr((gpuElem*)elements);
	gpuElem init = elements[0];
	gpuElem min = thrust::reduce(elem_ptr, elem_ptr+elemCount, init, min_gpu_elem());
	hipMalloc(result, storeElemSize);
	hipMemcpy(*result, &min, storeElemSize, hipMemcpyDeviceToDevice);

	return storeElemSize;
}
