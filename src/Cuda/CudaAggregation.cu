#include "hip/hip_runtime.h"
#include "CudaAggregation.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>


// HOW TO PRINT STH TO CONSOLE IN KERNEL
/*
// System includes
#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include "cuPrintf.cuh"

#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
                                  blockIdx.y*gridDim.x+blockIdx.x,\
                                  threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
                                  __VA_ARGS__)

// CUPRINTF("\tIdx: %d, tag: %d, metric: %d, val: %f, Value is:%d\n", idx, tag, elements[idx].metric, elements[idx].value, 1);
*/


// MIN AND MAX

struct min_gpu_elem
{
    __host__ __device__
        storeElement operator()(const storeElement &lhs, const storeElement &rhs) const
    {
    	return lhs.value < rhs.value ? lhs : rhs;
    }
};

struct max_gpu_elem
{
    __host__ __device__
        storeElement operator()(const storeElement &lhs, const storeElement &rhs) const
    {
    	return lhs.value < rhs.value ? rhs : lhs;
    }
};

size_t gpu_max(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	storeElement init;
	hipMemcpy(&init, elements, storeElemSize, hipMemcpyDeviceToHost);

	storeElement* max =
			new storeElement(thrust::reduce(elem_ptr, elem_ptr+elemCount, init, max_gpu_elem()));
	(*result) = max;

	return storeElemSize;
}

size_t gpu_min(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);
	storeElement init;
	hipMemcpy(&init, elements, storeElemSize, hipMemcpyDeviceToHost);

	storeElement* min =
			new storeElement(thrust::reduce(elem_ptr, elem_ptr+elemCount, init, min_gpu_elem()));
	(*result) = min;

	return storeElemSize;
}

// SUM AND AVERAGE

template <typename T>
struct sum_unary_op
{
	__host__ __device__
	float operator()(const T& x) const
	{
		return x.value;
	}
};

size_t gpu_sum(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	float init = 0.0f;
	sum_unary_op<storeElement> unary_op;

	results::sumResult* sum =
			new results::sumResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, thrust::plus<float>()));
	(*result) = sum;

	return sizeof(results::sumResult);
}

size_t gpu_average(ddj::store::storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	float init = 0.0f;
	sum_unary_op<storeElement> unary_op;

	results::averageResult* average =
			new results::averageResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, thrust::plus<float>()), elemCount);
	(*result) = average;

	return sizeof(results::averageResult);
}

// STD DEVIATION AND VARIANCE

template <typename T>
struct variance_unary_op
{
	__host__ __device__
	results::varianceResult operator()(const T& x) const
	{
		results::varianceResult result;
		result.count = 1;
		result.mean = x.value;
		result.M2 = 0;
		return result;
	}
};

struct variance_binary_op
    : public thrust::binary_function<const results::varianceResult&,
                                     const results::varianceResult&,
                                     results::varianceResult >
{
    __host__ __device__
    results::varianceResult operator()(const results::varianceResult& x, const results::varianceResult& y) const
    {
    	results::varianceResult result;

    	float count = x.count + y.count;
    	float delta = y.mean - x.mean;
    	float delta2 = delta * delta;
        result.count = count;
        result.mean = x.mean + delta * y.count / count;
        result.M2 = x.M2 + y.M2;
        result.M2 += delta2 * x.count * y.count / count;

        return result;
    }
};

size_t gpu_variance(storeElement* elements, size_t dataSize, void** result)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	thrust::device_ptr<storeElement> elem_ptr(elements);

	variance_unary_op<storeElement> unary_op;
	variance_binary_op binary_op;
	results::varianceResult init;

	results::varianceResult* variance =
			new results::varianceResult(thrust::transform_reduce(elem_ptr, elem_ptr+elemCount, unary_op, init, binary_op));
	(*result) = variance;

	return sizeof(results::varianceResult);
}

// TRUNK INTEGRAL

__global__ void calculate_trapezoid_fields(ddj::store::storeElement* elements, int count, float* result)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= count) return;

	ullint timespan = elements[idx+1].time - elements[idx].time;
	result[idx] = ( elements[idx].value + elements[idx+1].value ) * timespan / 2;
}

__global__ void sum_fields_in_trunks(float* fields, size_t elemSize, ddj::ullintPair* locations, int count, float* result)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= count) return;

	int i = locations[idx].first/elemSize;
	int end = locations[idx].second/elemSize;
	float sum = 0;
	for(; i<end; i++)
	{
		sum += fields[i];
	}
	result[idx] = sum;
}

__global__ void fill_integralResults(
		results::integralResult* result,
		storeElement* elements,
		size_t elemSize,
		float* integralSums,
		ddj::ullintPair* locations,
		int count)
{
	unsigned int i = blockIdx.x *blockDim.x + threadIdx.x;
	if(i >= count) return;

	result[i].integral = integralSums[i];
	int left = locations[i].first/elemSize;
	int right = locations[i].second/elemSize;
	result[i].left_value = elements[left].value;
	result[i].left_time= elements[left].time;
	result[i].right_value = elements[right].value;
	result[i].right_time= elements[right].time;
}

size_t gpu_trunk_integral(storeElement* elements, size_t dataSize, void** result,
		ddj::ullintPair* dataLocationInfo, int locationInfoCount)
{
	size_t storeElemSize = sizeof(storeElement);
	int elemCount = dataSize / storeElemSize;

	// ALLOCATE SPACE FOR RESULTS
	float* integralSums;
	hipMalloc(&integralSums, sizeof(float)*locationInfoCount);
	float* trapezoidFields;
	hipMalloc(&trapezoidFields, sizeof(float)*(elemCount-1));

	// CREATE TIME PERIODS VECTOR ON GPU
	thrust::device_vector<ddj::ullintPair> locations(dataLocationInfo, dataLocationInfo+locationInfoCount);

	// CALCULATE TRAPEZOID FIELDS
	int blocksPerGrid = (elemCount - 1 + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
	calculate_trapezoid_fields<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(elements, elemCount-1, trapezoidFields);
	hipDeviceSynchronize();

	// SUM UP FIELDS IN TRUNKS
	blocksPerGrid = (locationInfoCount + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
	sum_fields_in_trunks<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(
				trapezoidFields,
				sizeof(storeElement),
				locations.data().get(),
				locationInfoCount,
				integralSums);
	hipDeviceSynchronize();
	hipFree(trapezoidFields);

	// CREATE RESULT
	results::integralResult* integral = new results::integralResult[locationInfoCount];
	results::integralResult* integral_on_device;
	hipMalloc((void**)&integral_on_device, sizeof(results::integralResult)*locationInfoCount);
	fill_integralResults<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(
			integral_on_device,
			elements,
			storeElemSize,
			integralSums,
			locations.data().get(),
			locationInfoCount);
	hipMemcpy(integral, integral_on_device, sizeof(results::integralResult)*locationInfoCount, hipMemcpyDeviceToHost);
	hipFree(integral_on_device);
	hipFree(integralSums);

	// RETURN RESULT
	(*result)=integral;
	return locationInfoCount*sizeof(results::integralResult);
}
















