#include "hip/hip_runtime.h"
#include "CudaQuery.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>


// HOW TO PRINT STH TO CONSOLE IN KERNEL
// System includes
#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
                                  blockIdx.y*gridDim.x+blockIdx.x,\
                                  threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
                                  __VA_ARGS__)
// CUPRINTF("\tIdx: %d, tag: %d, metric: %d, val: %f, Value is:%d\n", idx, tag, elements[idx].metric, elements[idx].value, 1);

// TODO: Move this define to config
#define CUDA_THREADS_PER_BLOCK 256

typedef struct
{
	int32_t tag;
	int metric;
	ullint time;
	float value;
} gpuElem;

// TODO: Remove repeating code

__device__ bool isInside(ullint value, ddj::ullintPair* timePeriod)
{
	if(value >= timePeriod->first && value <= timePeriod->second) return true;
	else return false;
}

__global__ void cuda_produce_stencil_using_tag(
		ddj::store::storeElement* elements,
		int elemCount,
		int* tags,
		int tagsCount,
		int* stencil)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= elemCount) return;
	int32_t tag = elements[idx].tag;
	stencil[idx] = 0;
	while(tagsCount--)
	{
		if(tag == tags[tagsCount])
		{
			stencil[idx] = 1;
			return;
		}
	}
	return;
}

__global__ void cuda_produce_stencil_using_time(
		ddj::store::storeElement* elements,
		int elemCount,
		ddj::ullintPair* timePeriods,
		int timePeriodsCount,
		int* stencil)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= elemCount) return;
	ullint time = elements[idx].time;
	stencil[idx] = 0;
	while(timePeriodsCount--)
	{
		if(isInside(time, &timePeriods[timePeriodsCount]))
		{
			stencil[idx] = 1;
			return;
		}
	}
	return;
}

__global__ void cuda_produce_stencil_using_tag_and_time(
		ddj::store::storeElement* elements,
		int elemCount,
		int* tags,
		int tagsCount,
		ddj::ullintPair* timePeriods,
		int timePeriodsCount,
		int* stencil)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= elemCount) return;
	int32_t tag = elements[idx].tag;
	ullint time = elements[idx].time;
	stencil[idx] = 0;
	while(tagsCount--)
	{
		if(tag == tags[tagsCount])
		{
			while(timePeriodsCount--)
			{
				if(isInside(time, &timePeriods[timePeriodsCount]))
				{
					stencil[idx] = 1;
					return;
				}
			}
		}
	}
	return;
}

struct is_one
{
	__host__ __device__
	bool operator()(const int &x)
	{
		return x == 1;
	}
};



size_t gpu_filterData(ddj::store::storeElement* elements, size_t dataSize, ddj::store::storeQuery* query)
{
	// CREATE STENCIL
	int elemCount = dataSize/sizeof(ddj::store::storeElement);
	int* stencil;
	hipMalloc(&stencil, elemCount*sizeof(int));

	// FILL STENCIL
	int blocksPerGrid =(elemCount + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

	// CREATE TIME PERIODS VECTOR ON GPU
	thrust::device_vector<ddj::ullintPair> timePeriods(query->timePeriods.begin(), query->timePeriods.end());
	// CREATE TAGS VECTOR ON GPU
	thrust::device_vector<int> tags(query->tags.begin(), query->tags.end());

	// RUN STENCIL KERNEL
	int filterTags = query->tags.size();
	int filterTimePeriods = query->timePeriods.size();
	if(filterTags && filterTimePeriods)
	{
		cuda_produce_stencil_using_tag_and_time<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(
				elements,
				elemCount,
				tags.data().get(),
				tags.size(),
				timePeriods.data().get(),
				timePeriods.size(),
				stencil);
	} else if(filterTags){
		cuda_produce_stencil_using_tag<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(
				elements,
				elemCount,
				tags.data().get(),
				tags.size(),
				stencil);
	} else {
		cuda_produce_stencil_using_time<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(
				elements,
				elemCount,
				timePeriods.data().get(),
				timePeriods.size(),
				stencil);
	}
	hipDeviceSynchronize();

	// PARTITION ELEMENTS
	thrust::device_ptr<gpuElem> elem_ptr((gpuElem*)elements);
	thrust::device_ptr<int> stencil_ptr(stencil);

	thrust::partition(thrust::device, elem_ptr, elem_ptr+elemCount, stencil, is_one());

	// RETURN NUMBER OF ELEMENTS WITH TAG FROM QUERY'S TAGS
	return thrust::count(stencil_ptr, stencil_ptr+elemCount, 1) * sizeof(ddj::store::storeElement);
}
