#include "hip/hip_runtime.h"
#include "CudaQuery.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>

#define CUDA_THREADS_PER_BLOCK 256

typedef struct
{
	int tag;
	int metric;
	unsigned long long int time;
	float value;
} gpuElem;

__global__ void cuda_produce_stencil(ddj::store::storeElement* elements, int elemCount, int* tags, int tagsCount, int* stencil)
{
	unsigned int idx = blockIdx.x *blockDim.x + threadIdx.x;
	if(idx >= elemCount) return;
	int tag = elements[idx].tag;
	stencil[idx] = 0;
	while(tagsCount--)
	{
		if(tag == tags[tagsCount])
		{
			stencil[idx] = 1;
			return;
		}
	}

}

struct is_one
{
	__host__ __device__
	bool operator()(const int &x)
	{
		return x == 1;
	}
};

size_t gpu_filterData(ddj::store::storeElement* elements, int elemCount, ddj::store::storeQuery* query)
{
	// CREATE STENCIL
	int* stencil;
	hipMalloc(&stencil, elemCount*sizeof(int));

	// CREATE TAGS VECTOR ON GPU
	thrust::device_vector<int> tags(query->tags.begin(), query->tags.end());

	// FILL STENCIL
	int blocksPerGrid =(elemCount + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
	cuda_produce_stencil<<<blocksPerGrid, CUDA_THREADS_PER_BLOCK>>>(elements, elemCount, tags.data().get(), tags.size(), stencil);

	// PARTITION ELEMENTS
	thrust::device_ptr<gpuElem> elem_ptr((gpuElem*)elements);
	thrust::device_ptr<int> stencil_ptr(stencil);

	thrust::partition(thrust::device, elem_ptr, elem_ptr+elemCount, stencil, is_one());

	// RETURN NUMBER OF ELEMENTS WITH TAG FROM QUERY'S TAGS
	return thrust::count(stencil_ptr, stencil_ptr+elemCount, 1);
}
